#include "LBMSRT.cuh"
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

int main()
{
	const int Nstep = 8000;
	int savepoint = 1000;
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	LBMpara params;
	params.Nx = 40;
	params.Ny = 40;
	params.Nz = 960;
	params.rho0 = 1.0;
	params.ux0 = 0.0;
	params.uy0 = 0.0;
	params.uz0 = 0.2;

	LBMgpu lbm;

	cout << "Simulation start!" << endl;
	lbm.init(params);

	cout << "Loop start!" << endl;
	hipEventRecord(start, 0);

	for (int step = 1; step <= Nstep; step++)
	{
		lbm.feq();
		lbm.rate_strain();
		lbm.collision();
		lbm.swap();
		lbm.boundary();		
		lbm.calrhov();

		if (step % savepoint == 0)
			lbm.output(step);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cout << "Loop time is: " << time << endl;

	hipEventDestroy(start);
	hipEventDestroy(stop);
	lbm.freemem();

	hipDeviceReset();
	system("pause");
	return 0;
}